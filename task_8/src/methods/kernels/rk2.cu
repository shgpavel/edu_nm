
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void rk2_cuda(double h, double A, double B,
												 double a21, double b1, double b2,
												 unsigned num_steps, double *y_initial,
												 double *y_values) {
    
	int system_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	double *y_current = y_initial + 2 * system_idx;
	double *y_all_steps = y_values + system_idx;
	
	double y1 = y_current[0];
	double y2 = y_current[1];

	y_all_steps[0] = y1;
	y_all_steps[num_steps * 2 * gridDim.x] = y2;
	
	for (unsigned step = 1; step < num_steps; ++step) {
		double k1_y1 = A * y2;
		double k1_y2 = -B * y1;
    
		double y_temp1 = y1 + a21 * h * k1_y1;
		double y_temp2 = y2 + a21 * h * k1_y2;
		
		double k2_y1 = A * y_temp2;
		double k2_y2 = -B * y_temp1;
    
		y1 = y1 + h * (b1 * k1_y1 + b2 * k2_y1);
		y2 = y2 + h * (b1 * k1_y2 + b2 * k2_y2);
    
		y_all_steps[step * 2 * gridDim.x] = y1;
		y_all_steps[step * 2 * gridDim.x + 1] = y2;
	}
}
